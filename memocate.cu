/* memocate.cu: The Trivial Memory Allocator in CUDA */
/* Disclaimer: I am not responsible for anything that happens. You are warned */
/* Author: TD */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>


int main(void)
{
	int size = 2147483647;
	char *a = malloc(size);
	char *d_a;
	hipMalloc((void **)&d_a,size);
	hipMemcpy(d_a,a,size,cudaMemcpyHosttovice);
	hipFree(d_a);
	free(a); 
	return 0;

}
